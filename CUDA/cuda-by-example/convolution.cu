/*
 * @FileName   : basis.cu 
 * @CreateAt   : 2021/12/6
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: 
 */
#include <iostream>

#include <hip/hip_runtime.h>


#define WIDTH 1920
#define HEIGHT 1080
#define real float

static void HandleError(hipError_t err, const char *file=__FILE__, int line=__LINE__) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

int getThreadNum() {
    hipDeviceProp_t prop;
    int count;

    HandleError( hipGetDeviceCount(&count) );
    printf("GPU device count is %d\n", count);
    HandleError( hipGetDeviceProperties(&prop, 0) );
    printf("Warp size is %d\n", prop.warpSize);
    printf("Max thread number per block is %d\n", prop.maxThreadsPerBlock);
    printf("Max block number per MultiProcessor is %d\n", prop.maxBlocksPerMultiProcessor);
    printf("The dimensions of grid is (%d, %d, %d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

__global__
void conv(real *img, real *kernel, real *result, int kernel_dim) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    if (id < WIDTH * HEIGHT) {
        int row = id / WIDTH;
        int col = id % WIDTH;
        for (int i = 0; i < kernel_dim; ++ i) {
            for (int j = 0; j < kernel_dim; ++ j) {
                real img_val = 0;
                int curr_row = row - kernel_dim / 2 + i;
                int curr_col = col - kernel_dim / 2 + j;
                if (0 <= curr_row && curr_row < HEIGHT && 0 <= curr_col && curr_col < WIDTH) {
                    img_val = img[curr_row * WIDTH + curr_col];
                }
                result[id] += kernel[i * kernel_dim + j] * img_val;
            }
        }
    }
}

int main() {

    /* INITIALIZE */

    real *img = new real[WIDTH * HEIGHT];
    for (int row = 0; row < HEIGHT; ++ row) {
        for (size_t col = 0; col < WIDTH; ++ col) {
            img[col + row * WIDTH] = static_cast<real>((col + row) % 256);
        }
    }

    int kernel_dim = 3;
    int kernel_size = kernel_dim * kernel_dim;
    real *kernel = new real[kernel_size];
    for (int i = 0; i < kernel_size; ++i) {
        kernel[i] = static_cast<real>(i % kernel_dim - 1); // each row is [-1 0 1]
    }


    real *img_d, *kernel_d, *result_d;
    HandleError( hipMalloc((void**)&img_d, WIDTH * HEIGHT * sizeof(real)) );
    HandleError( hipMalloc((void**)&kernel_d, kernel_size * sizeof(real)) );
    HandleError( hipMalloc((void**)&result_d, WIDTH * HEIGHT * sizeof(real)) );

    HandleError( hipMemcpy(img_d, img, WIDTH * HEIGHT * sizeof(real), hipMemcpyHostToDevice) );
    HandleError( hipMemcpy(kernel_d, kernel, kernel_size * sizeof(real), hipMemcpyHostToDevice) );

    /* COMPUTING */

    int thread_num = getThreadNum();
    int block_num = ( WIDTH * HEIGHT + thread_num - 1 ) / thread_num;
    conv<<<block_num, thread_num>>>(img_d, kernel_d, result_d, kernel_dim);

    real *result = new real[WIDTH * HEIGHT];
    HandleError( hipMemcpy(result, result_d, WIDTH * HEIGHT * sizeof(real), hipMemcpyDeviceToHost) );

    /* DISPLAY */

    // print image
    printf("\nimage\n===========\n");
    for (int row = 0; row < 10; ++ row) {
        for (int col = 0; col < 10; ++ col) {
            printf("%3.0f", img[col + row * WIDTH]);
        }
        printf("\n");
    }

    // print kernel
    printf("\nkernel\n===========\n");
    for (int row = 0; row < kernel_dim; ++ row) {
        for (int col = 0; col < kernel_dim; ++ col) {
            printf("%3.0f", kernel[col + row * kernel_dim]);
        }
        printf("\n");
    }

    // print result
    printf("\nresult\n===========\n");
    for (size_t row = 0; row < 10; ++ row) {
        for (size_t col = 0; col < 10; ++ col) {
            printf("%3.0f", result[col + row * WIDTH]);
        }
        printf("\n");
    }

    hipFree(img_d);
    hipFree(kernel_d);
    hipFree(result_d);

    free(img);
    free(kernel);
    free(result);

    return 0;
}

